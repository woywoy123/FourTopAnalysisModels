#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


template <typename scalar_t>
__device__ __forceinline__ void AggregatePath(
		scalar_t* Output, 
		const scalar_t* FourVec, 
		const scalar_t* Selector)
{

	(*Output) += (*FourVec)*(*Selector);  
}

template <typename scalar_t>
__global__ void SelectorKernel(
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> PTH, 
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> FV, 
	torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> Pmu,
	const size_t cmbi_l, 
	const size_t nodes)
{
	const int nd = blockIdx.y;
	const int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < cmbi_l)
	{
		for (unsigned int i = 0; i < nodes; i++)
		{
			AggregatePath(&(Pmu[index][nd]), &(FV[i][nd]), &(PTH[index][i])); 
		}
	}
}

torch::Tensor PathVectorGPU(torch::Tensor AdjMatrix, torch::Tensor FourVector)
{
	torch::TensorOptions options = torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA);
	const int cmbi_l = AdjMatrix.size(0);
	const int nodes = AdjMatrix.size(1);
	const int threads = 1024;
	const dim3 blocks((cmbi_l + threads -1) / threads, 4);
	
	torch::Tensor Pmu = torch::zeros({cmbi_l, 4}, options);
	FourVector = FourVector.to(options);
	AdjMatrix = AdjMatrix.to(options);

	AT_DISPATCH_FLOATING_TYPES(torch::kFloat, "SelectorKernel", ([&]
	{
		SelectorKernel<scalar_t><<<blocks, threads>>>(
			AdjMatrix.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(), 
			FourVector.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			cmbi_l, nodes
		);
	}));
	return Pmu;
}


template <typename scalar_t>
__global__ void NodeSelectorKernel(
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> PTH, 
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> FV, 
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> NodeIndex, 
	torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> Pmu,
	const size_t cmbi_l, 
	const size_t nodes)
{
	const int adj = blockIdx.y*blockDim.y + threadIdx.y;
	const int index = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (index < cmbi_l)
	{
		for (unsigned int i = 0; i < nodes; i++)
		{
			//const int edge_index = index - NodeIndex[index][0]*(PTH.size(0)-1) + i; 
			//AggregatePath(&(Pmu[index][nd]), &(FV[edge_index][nd]), &(PTH[adj_index][i])); 
			
			printf("%d %d %d\n", index, PTH[adj][i], adj); 
			Pmu[index][0] += PTH[adj][i];
		}
	}
}

torch::Tensor IncomingEdgeVectorGPU(torch::Tensor AdjMatrix, torch::Tensor IncomingEdges, torch::Tensor Index)
{

	torch::TensorOptions options = torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA); 
	const int edges = IncomingEdges.size(0); 
	const int adj = AdjMatrix.size(0);
	const int nodes = AdjMatrix.size(1);

	torch::Tensor Pmu = torch::zeros({adj*nodes, 4}, options); 
	const int threads = 1024;
	const dim3 blocks((Pmu.size(0) + threads -1) / threads, 4);

	Index = Index.to(options);
	IncomingEdges = IncomingEdges.to(options);
	AdjMatrix = IncomingEdges.to(options);
	AT_DISPATCH_FLOATING_TYPES(torch::kFloat, "NodeSelectorKernel", ([&]
	{
		NodeSelectorKernel<scalar_t><<<blocks, threads>>>(
			AdjMatrix.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(), 
			IncomingEdges.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Index.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu.size(0), nodes
		);
	})); 
	return Pmu; 

}
