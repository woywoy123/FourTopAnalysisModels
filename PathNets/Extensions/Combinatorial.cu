#include "hip/hip_runtime.h"
#include <sys/types.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

template <typename scalar_t>
__device__ __forceinline__ void AggregatePath(
		scalar_t* Output, 
		const scalar_t* FourVec, 
		const scalar_t* Selector)
{
	(*Output) += (*FourVec)*(*Selector);  
}

template <typename scalar_t>
__global__ void SelectorKernel(
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> PTH, 
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> FV, 
	torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> Pmu,
	const size_t cmbi_l, 
	const size_t nodes)
{
	const int nd = blockIdx.y;
	const int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < cmbi_l)
	{
		for (unsigned int i = 0; i < nodes; i++)
		{
			AggregatePath(
				&(Pmu[index][nd]), 
				&(FV[i][nd]), 
				&(PTH[index][i])); 
		}
	}
}

torch::Tensor PathVectorGPU(torch::Tensor AdjMatrix, torch::Tensor FourVector)
{
	torch::TensorOptions options = torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA);
	const int cmbi_l = AdjMatrix.size(0);
	const int nodes = AdjMatrix.size(1);
	const int threads = 1024;
	const dim3 blocks((cmbi_l + threads -1) / threads, 4);
	
	torch::Tensor Pmu = torch::zeros({cmbi_l, 4}, options);
	FourVector = FourVector.to(options);
	AdjMatrix = AdjMatrix.to(options);

	AT_DISPATCH_FLOATING_TYPES(torch::kFloat, "SelectorKernel", ([&]
	{
		SelectorKernel<scalar_t><<<blocks, threads>>>(
			AdjMatrix.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(), 
			FourVector.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			cmbi_l, 
			nodes
		);
	}));
	return Pmu;
}



template <typename scalar_t>
__device__ __forceinline__ void AggregatePath(
		scalar_t* Output, 
		scalar_t* IncomingEdgeAdj, 
		const scalar_t* IncomingEdgeVector, 
		const scalar_t* AdjMat, 
		const scalar_t* edgeindex,
		const int* node, 
		const int* adj)
{
	if ((*node) != (*edgeindex) || (*AdjMat) == 0){return;}
	(*Output) += (*IncomingEdgeVector); 
	(*IncomingEdgeAdj) = (*adj);
}


template <typename scalar_t>
__global__ void NodeSelectorKernel(
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> PTH, 
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> FV, 
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> NodeIndex, 
	torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> Pmu_adj,
	torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> Pmu,
	const size_t cmbi_l, 
	const size_t nodes)
{
	const int adj = blockIdx.x*blockDim.x + threadIdx.x;
	const int nod = blockIdx.y;
	const int dim = blockIdx.z;
	const int index = adj + nod*PTH.size(0);

	if (index < cmbi_l && adj < PTH.size(0))
	{
		for (unsigned int i = 0; i < nodes; i++)
		{
			AggregatePath(
				&(Pmu[index][dim]),
				&(Pmu_adj[index][0]),
				&(FV[i+nod*nodes][dim]), 
				&(PTH[adj][i]), 
				&(NodeIndex[i + nod*nodes][0]), 
				&nod,
				&adj); 
		}
	}
}

std::vector<torch::Tensor> IncomingEdgeVectorGPU(torch::Tensor AdjMatrix, torch::Tensor IncomingEdges, torch::Tensor Index)
{

	torch::TensorOptions options = torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA); 
	const int adj = AdjMatrix.size(0);
	const int nodes = AdjMatrix.size(1);
	const int threads = 1024;
	const dim3 blocks((adj+threads-1)/threads, nodes, 4);

	torch::Tensor Pmu = torch::zeros({adj*nodes, 4}, options); 
	torch::Tensor Pmu_adj = torch::zeros({adj*nodes, 1}, options); 
	Index = Index.to(options);
	IncomingEdges = IncomingEdges.to(options);
	AdjMatrix = AdjMatrix.to(options);

	AT_DISPATCH_FLOATING_TYPES(torch::kFloat, "NodeSelectorKernel", ([&]
	{
		NodeSelectorKernel<scalar_t><<<blocks, threads>>>(
			AdjMatrix.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(), 
			IncomingEdges.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Index.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu_adj.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu.size(0), 
			nodes
		);
	})); 

	return {Pmu, Pmu_adj}; 
}



template <typename scalar_t>
__global__ void CombinatorialKernel(
	torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> Combi,
	const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> t,
	const size_t nodes)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < t.size(0))
	{
		int n = t[i];
		for (unsigned int l = 0; l < nodes; l++){ Combi[i][l] = n & 1; n >>= 1;}
	}
}

torch::Tensor PathCombinatorialGPU(const int nodes, torch::Tensor t)
{
	const int threads = 1024;
	const dim3 blocks((t.size(0) + threads -1)/threads, 1); 
		
	std::cout << "!!!! WARNING !!!!! NOT WORKING AS EXPECTED!" << std::endl;
	torch::TensorOptions options = torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA);
	torch::Tensor Combi = torch::zeros({t.size(0), nodes}, options);
	AT_DISPATCH_FLOATING_TYPES(torch::kFloat, "CombinatorialKernel", ([&]
	{
		CombinatorialKernel<scalar_t><<<blocks, threads>>>(
			Combi.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			t.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
			nodes
		);
	}));
	
	Combi = Combi.to(options.dtype(torch::kInt));

	return Combi; 
}
