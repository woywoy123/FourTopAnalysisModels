#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>




template <typename scalar_t>
__global__ void Kernel(
		const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> PTH, 
		const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> FV, 




torch::Tensor PathMassGPU(torch::Tensor AdjMatrix, torch::Tensor FourVector)
{
	const int l = AdjMatrix.size(0);
	





	return AdjMatrix;

}
