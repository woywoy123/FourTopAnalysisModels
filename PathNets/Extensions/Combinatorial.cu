#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

template <typename scalar_t>
__device__ __forceinline__ void AggregatePath(
		scalar_t* Output, 
		const scalar_t* FourVec, 
		const scalar_t* Selector)
{
	(*Output) += (*FourVec)*(*Selector);  
}

template <typename scalar_t>
__global__ void SelectorKernel(
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> PTH, 
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> FV, 
	torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> Pmu,
	const size_t cmbi_l, 
	const size_t nodes)
{
	const int nd = blockIdx.y;
	const int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < cmbi_l)
	{
		for (unsigned int i = 0; i < nodes; i++)
		{
			AggregatePath(
				&(Pmu[index][nd]), 
				&(FV[i][nd]), 
				&(PTH[index][i])); 
		}
	}
}

torch::Tensor PathVectorGPU(torch::Tensor AdjMatrix, torch::Tensor FourVector)
{
	torch::TensorOptions options = torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA);
	const int cmbi_l = AdjMatrix.size(0);
	const int nodes = AdjMatrix.size(1);
	const int threads = 1024;
	const dim3 blocks((cmbi_l + threads -1) / threads, 4);
	
	torch::Tensor Pmu = torch::zeros({cmbi_l, 4}, options);
	FourVector = FourVector.to(options);
	AdjMatrix = AdjMatrix.to(options);

	AT_DISPATCH_FLOATING_TYPES(torch::kFloat, "SelectorKernel", ([&]
	{
		SelectorKernel<scalar_t><<<blocks, threads>>>(
			AdjMatrix.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(), 
			FourVector.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			cmbi_l, 
			nodes
		);
	}));
	return Pmu;
}



template <typename scalar_t>
__device__ __forceinline__ void AggregatePath(
		scalar_t* Output, 
		const scalar_t* IncomingEdgeVector, 
		const scalar_t* AdjMat, 
		const scalar_t* edgeindex,
		const int* node)
{
	if ((*node) != (*edgeindex) || (*AdjMat) == 0){return;}
	(*Output) += (*IncomingEdgeVector);  
}


template <typename scalar_t>
__global__ void NodeSelectorKernel(
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> PTH, 
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> FV, 
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> NodeIndex, 
	torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> Pmu,
	const size_t cmbi_l, 
	const size_t nodes)
{
	const int adj = blockIdx.x*blockDim.x + threadIdx.x;
	const int nod = blockIdx.y;
	const int dim = blockIdx.z;
	const int index = adj + nod*PTH.size(0);

	if (index < cmbi_l && adj < PTH.size(0))
	{
		for (unsigned int i = 0; i < nodes; i++)
		{
			AggregatePath(
				&(Pmu[index][dim]), 
				&(FV[i+nod*nodes][dim]), 
				&(PTH[adj][i]), 
				&(NodeIndex[i + nod*nodes][0]), 
				&nod); 
		}
	}
}

torch::Tensor IncomingEdgeVectorGPU(torch::Tensor AdjMatrix, torch::Tensor IncomingEdges, torch::Tensor Index)
{

	torch::TensorOptions options = torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA); 
	const int adj = AdjMatrix.size(0);
	const int nodes = AdjMatrix.size(1);
	const int threads = 1024;
	const dim3 blocks((adj+threads-1)/threads, nodes, 4);

	torch::Tensor Pmu = torch::zeros({adj*nodes, 4}, options); 
	Index = Index.to(options);
	IncomingEdges = IncomingEdges.to(options);
	AdjMatrix = AdjMatrix.to(options);

	AT_DISPATCH_FLOATING_TYPES(torch::kFloat, "NodeSelectorKernel", ([&]
	{
		NodeSelectorKernel<scalar_t><<<blocks, threads>>>(
			AdjMatrix.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(), 
			IncomingEdges.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Index.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu.size(0), 
			nodes
		);
	})); 

	return Pmu; 
}



//template <typename scalar_t>
//__global__ void CombinatorialKernel(
//
//


// Need to continue here. Write a non recursive binary combinatorial....
torch::Tensor PathCombinatorialGPU(int n, int k, int len)
{
	const int threads = 1024; 
	const dim3 blocks((n + threads -1)/threads, k); 

	torch::TensorOptions options = torch::TensorOptions().dtype(torch::kInt).device(torch::kCUDA);
	torch::Tensor Combi = torch::zeros({len, n}, options);
	torch::Tensor MSK = torch::pow(2, torch::arange(n, options));

	return torch::tensor({1, 2}, options); 
}
