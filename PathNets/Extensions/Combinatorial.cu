#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

template <typename scalar_t>
__device__ __forceinline__ scalar_t VecCalc(scalar_t fv, scalar_t sw)
{
	return fv*sw; 
}


template <typename scalar_t>
__global__ void MassKernel(
	scalar_t* __restrict__ Mass,
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> PTH, 
	const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> FV, 
	torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> Pmu,
	const size_t cmbi_l, 
	const size_t nodes)
{
	const int nd = blockIdx.y;
	const int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < nodes)
	{
		printf("%d, %d\n", nd, index);
		Pmu[nd][0] += FV[index][0];
		Pmu[nd][1] += FV[index][1];
		Pmu[nd][2] += FV[index][2];
		Pmu[nd][3] += FV[index][3];
	}

}
torch::Tensor PathMassGPU(torch::Tensor AdjMatrix, torch::Tensor FourVector)
{
	torch::TensorOptions options = torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA);
	const int cmbi_l = AdjMatrix.size(0);
	const int nodes = AdjMatrix.size(1);
	std::cout << nodes << " " << cmbi_l << std::endl;
	const int threads = nodes;
	const dim3 blocks((nodes + threads -1) / threads, cmbi_l);
	
	torch::Tensor Mass = torch::zeros({cmbi_l, 1}, options);
	torch::Tensor Pmu = torch::zeros({cmbi_l, 4}, options);
	FourVector = FourVector.to(options);
	AdjMatrix = AdjMatrix.to(options);

	AT_DISPATCH_FLOATING_TYPES(torch::kFloat, "MassKernel", ([&]
	{
		MassKernel<scalar_t><<<blocks, threads>>>(
			Mass.data_ptr<scalar_t>(),
			AdjMatrix.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(), 
			FourVector.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			Pmu.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
			cmbi_l, nodes
		);
	}));
	return Pmu;

}
